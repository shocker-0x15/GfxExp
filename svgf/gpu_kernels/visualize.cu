#include "hip/hip_runtime.h"
﻿#define PURE_CUDA
#include "../svgf_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void debugVisualize(
    BufferToDisplay bufferToDisplay,
    float motionVectorOffset, float motionVectorScale,
    uint32_t numFilteringStages)
{
    const uint2 launchIndex = make_uint2(
        blockDim.x * blockIdx.x + threadIdx.x,
        blockDim.y * blockIdx.y + threadIdx.y);
    const int2 pix = make_int2(launchIndex.x, launchIndex.y);
    const int2 imageSize = plp.s->imageSize;
    if (launchIndex.x >= imageSize.x ||
        launchIndex.y >= imageSize.y)
        return;

    const uint32_t curBufIdx = plp.f->bufferIndex;
    const StaticPipelineLaunchParameters::TemporalSet &staticTemporalSet =
        plp.s->temporalSets[curBufIdx];
    const PerFramePipelineLaunchParameters::TemporalSet &perFrameTemporalSet =
        plp.f->temporalSets[curBufIdx];

    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    switch (bufferToDisplay) {
    case BufferToDisplay::NoisyBeauty: {
        const Lighting_Variance lighting_var =
            plp.s->lighting_variance_buffers[0].read(pix);
        RGB lighting = lighting_var.noisyLighting;
        if (plp.f->modulateAlbedo) {
            const Albedo albedo = plp.s->albedoBuffer.read(pix);
            lighting *= albedo.dhReflectance;
        }
        color = make_float4(lighting.toNative(), 1.0f);
        break;
    }
    case BufferToDisplay::Variance: {
        const Lighting_Variance lighting_var =
            plp.s->lighting_variance_buffers[0].read(pix);
        const float stdDev = std::sqrt(lighting_var.variance);
        color = make_float4(make_float3(stdDev), 1.0f);
        break;
    }
    case BufferToDisplay::FilteredVariance: {
        const Lighting_Variance filtered_lighting_var =
            plp.s->lighting_variance_buffers[numFilteringStages % 2].read(pix);
        const float stdDev = std::sqrt(filtered_lighting_var.variance);
        color = make_float4(make_float3(stdDev), 1.0f);
        break;
    }
    case BufferToDisplay::Albedo: {
        const Albedo albedo = plp.s->albedoBuffer.read(pix);
        color = make_float4(albedo.dhReflectance.toNative(), 1.0f);
        break;
    }
    case BufferToDisplay::Normal: {
        const GBuffer1Elements gb1Elems = perFrameTemporalSet.GBuffer1.read(glPix(pix));
        color = make_float4((0.5f * gb1Elems.normalInWorld + Normal3D(0.5f)).toNative(), 1.0f);
        break;
    }
    case BufferToDisplay::MotionVector: {
        const GBuffer2Elements gb2Elems = perFrameTemporalSet.GBuffer2.read(glPix(pix));
        const Point2D curScreenPos = Point2D(pix.x + 0.5f, pix.y + 0.5f) / Point2D(imageSize.x, imageSize.y);
        const Point2D prevScreenPos = gb2Elems.prevScreenPos;
        const Vector2D motionVector = Vector2D(imageSize.x, imageSize.y) * (curScreenPos - prevScreenPos);
        color = make_float4(
            clamp(motionVectorScale * motionVector.x + motionVectorOffset, 0.0f, 1.0f),
            clamp(motionVectorScale * motionVector.y + motionVectorOffset, 0.0f, 1.0f),
            motionVectorOffset, 1.0f);
        break;
    }
    case BufferToDisplay::SampleCount: {
        const MomentPair_SampleInfo momentPair_sampleInfo =
            staticTemporalSet.momentPair_sampleInfo_buffer.read(pix);
        const float value = min(momentPair_sampleInfo.sampleInfo.count / 255.0f, 1.0f);
        color = make_float4(make_float3(value), 1.0f);
        break;
    }
    default:
        Assert_ShouldNotBeCalled();
        break;
    }

    plp.f->debugVisualizeBuffer.write(glPix(pix), color);
}
