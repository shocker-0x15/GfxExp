#include "hip/hip_runtime.h"
﻿#include "../neural_radiance_caching_shared.h"

using namespace shared;

CUDA_DEVICE_KERNEL void RT_AH_NAME(visibility)() {
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set(&visibility);
}



CUDA_DEVICE_FUNCTION CUDA_INLINE void convertToPolar(const Vector3D &dir, float* phi, float* theta) {
    float z = std::fmin(std::fmax(dir.z, -1.0f), 1.0f);
    *theta = std::acos(z);
    *phi = std::atan2(dir.y, dir.x);
}

CUDA_DEVICE_FUNCTION CUDA_INLINE void createRadianceQuery(
    const Point3D &positionInWorld, const Normal3D &normalInWorld, const Vector3D &scatteredDirInWorld,
    float roughness, const RGB &diffuseReflectance, const RGB &specularReflectance,
    RadianceQuery* query)
{
    float phi, theta;
    query->position = plp.s->sceneAABB->normalize(positionInWorld);
    convertToPolar(Vector3D(normalInWorld), &phi, &theta);
    query->normal_phi = phi;
    query->normal_theta = theta;
    convertToPolar(scatteredDirInWorld, &phi, &theta);
    query->vOut_phi = phi;
    query->vOut_theta = theta;
    query->roughness = 1 - std::exp(-roughness);
    query->diffuseReflectance = diffuseReflectance;
    query->specularReflectance = specularReflectance;
}

static constexpr bool useSolidAngleSampling = false;

CUDA_DEVICE_FUNCTION CUDA_INLINE RGB performNextEventEstimation(
    const Point3D &shadingPoint, const Vector3D &vOutLocal, const ReferenceFrame &shadingFrame,
    const BSDF &bsdf, PCG32RNG &rng)
{
    float uLight = rng.getFloat0cTo1o();
    bool selectEnvLight = false;
    float probToSampleCurLightType = 1.0f;
    if (plp.s->envLightTexture && plp.f->enableEnvLight) {
        if (plp.s->lightInstDist.integral() > 0.0f) {
            if (uLight < probToSampleEnvLight) {
                probToSampleCurLightType = probToSampleEnvLight;
                uLight /= probToSampleCurLightType;
                selectEnvLight = true;
            }
            else {
                probToSampleCurLightType = 1.0f - probToSampleEnvLight;
                uLight = (uLight - probToSampleEnvLight) / probToSampleCurLightType;
            }
        }
        else {
            selectEnvLight = true;
        }
    }
    LightSample lightSample;
    float areaPDensity;
    sampleLight<useSolidAngleSampling>(
        shadingPoint,
        uLight, selectEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &lightSample, &areaPDensity);
    areaPDensity *= probToSampleCurLightType;

    Vector3D shadowRay = lightSample.atInfinity ?
        Vector3D(lightSample.position) :
        (lightSample.position - shadingPoint);
    const float dist2 = shadowRay.sqLength();
    shadowRay /= std::sqrt(dist2);
    const Vector3D vInLocal = shadingFrame.toLocal(shadowRay);
    const float lpCos = std::fabs(dot(shadowRay, lightSample.normal));
    float bsdfPDensity = bsdf.evaluatePDF(vOutLocal, vInLocal) * lpCos / dist2;
    if (!stc::isfinite(bsdfPDensity))
        bsdfPDensity = 0.0f;
    const float lightPDensity = areaPDensity;
    const float misWeight = pow2(lightPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
    RGB ret(0.0f);
    if (areaPDensity > 0.0f)
        ret = performDirectLighting<PathTracingRayType, true>(
            shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) * (misWeight / areaPDensity);
    //if (!allFinite(ret)) {
    //    printf("mis: %g / %g, p:(%g, %g, %g), v:(%g, %g, %g)\n",
    //           misWeight, areaPDensity,
    //           shadingPoint.x, shadingPoint.y, shadingPoint.z,
    //           vOutLocal.x, vOutLocal.y, vOutLocal.z);
    //}

    return ret;
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_raygen_generic() {
    const uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    const uint32_t bufIdx = plp.f->bufferIndex;

    const GBuffer0Elements gb0Elems = plp.s->GBuffer0[bufIdx].read(launchIndex);
    const uint32_t instSlot = gb0Elems.instSlot;
    const float bcB = decodeBarycentric(gb0Elems.qbcB);
    const float bcC = decodeBarycentric(gb0Elems.qbcC);

    const PerspectiveCamera &camera = plp.f->camera;

    uint32_t linearTileIndex;
    bool isTrainingPath;
    bool isUnbiasedTrainingTile;
    if constexpr (useNRC) {
        const uint2 tileSize = *plp.s->tileSize[bufIdx];
        const uint32_t numPixelsInTile = tileSize.x * tileSize.y;

        // JP: 動的サイズのタイルごとに1つトレーニングパスを選ぶ。
        // EN: choose a training path for each dynamic-sized tile.
        const uint2 localIndex = launchIndex % tileSize;
        const uint32_t localLinearIndex = localIndex.y * tileSize.x + localIndex.x;
        isTrainingPath = (localLinearIndex + *plp.s->offsetToSelectTrainingPath) % numPixelsInTile == 0;

        const uint2 numTiles = (plp.s->imageSize + tileSize - 1) / tileSize;
        const uint2 tileIndex = launchIndex / tileSize;
        linearTileIndex = tileIndex.y * numTiles.x + tileIndex.x;

        // JP: トレーニングパスの16本に1本はセルフトレーニングを使用しないUnbiasedパスとする。
        // EN: Make one path out of every 16 training paths not use self-training and unbiased.
        const uint2 tileGroupSize = make_uint2(4, 4);
        const uint2 localTileIndex = tileIndex % tileGroupSize;
        const uint32_t localLinearTileIndex = localTileIndex.y * tileGroupSize.x + localTileIndex.x;
        isUnbiasedTrainingTile = (localLinearTileIndex + *plp.s->offsetToSelectUnbiasedTile) % 16 == 0;
    }
    else {
        (void)linearTileIndex;
        (void)isTrainingPath;
        (void)isUnbiasedTrainingTile;
    }

    const bool useEnvLight = plp.s->envLightTexture && plp.f->enableEnvLight;
    RGB contribution(0.001f, 0.001f, 0.001f);
    bool renderingPathEndsWithCache = false;
    uint32_t pathLength = 1;
    if (instSlot != 0xFFFFFFFF) {
        const uint32_t geomInstSlot = gb0Elems.geomInstSlot;
        const InstanceData &inst = plp.s->instanceDataBufferArray[bufIdx][instSlot];
        const GeometryInstanceData &geomInst = plp.s->geometryInstanceDataBuffer[geomInstSlot];
        Point3D positionInWorld;
        Normal3D geometricNormalInWorld;
        Normal3D shadingNormalInWorld;
        Vector3D texCoord0DirInWorld;
        Point2D texCoord;
        computeSurfacePoint(
            inst, geomInst,
            gb0Elems.primIndex, bcB, bcC,
            &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
            &geometricNormalInWorld, &texCoord);

        RGB alpha(1.0f);
        const float initImportance = sRGB_calcLuminance(alpha);
        PCG32RNG rng = plp.s->rngBuffer.read(launchIndex);

        // JP: 最初の交点におけるシェーディング。
        // EN: Shading on the first hit.
        Vector3D vIn;
        float dirPDensity;
        float primaryPathSpread;
        RGB localThroughput;
        uint32_t trainDataIndex;
        {
            const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

            Vector3D vOut = camera.position - positionInWorld;
            const float primaryDist2 = vOut.sqLength();
            vOut /= std::sqrt(primaryDist2);
            const float primaryDotVN = dot(vOut, geometricNormalInWorld);
            const float frontHit = primaryDotVN >= 0.0f ? 1.0f : -1.0f;
            // Offsetting assumes BRDF.
            positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);

            if constexpr (useNRC)
                primaryPathSpread = primaryDist2 / (4 * pi_v<float> * std::fabs(primaryDotVN));

            ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
            if (plp.f->enableBumpMapping) {
                const Normal3D modLocalNormal = mat.readModifiedNormal(mat.normal, mat.normalDimInfo, texCoord, 0.0f);
                applyBumpMapping(modLocalNormal, &shadingFrame);
            }
            const Vector3D vOutLocal = shadingFrame.toLocal(vOut);

            // JP: 光源を直接見ている場合の寄与を蓄積。
            // EN: Accumulate the contribution from a light source directly seeing.
            contribution = RGB(0.0f);
            if (vOutLocal.z > 0 && mat.emittance) {
                const float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
                const RGB emittance(getXYZ(texValue));
                contribution += alpha * emittance / pi_v<float>;
            }

            BSDF bsdf;
            bsdf.setup(mat, texCoord, 0.0f);

            // Next event estimation (explicit light sampling) on the first hit.
            const RGB directContNEE = performNextEventEstimation(
                positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
            contribution += alpha * directContNEE;

            // generate a next ray.
            Vector3D vInLocal;
            localThroughput = bsdf.sampleThroughput(
                vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
                &vInLocal, &dirPDensity);
            alpha *= localThroughput;
            vIn = shadingFrame.fromLocal(vInLocal);

            if constexpr (useNRC) {
                // JP: 訓練データエントリーの確保。
                // EN: Allocate a training data entry.
                if (isTrainingPath) {
                    trainDataIndex = atomicAdd(plp.s->numTrainingData[bufIdx], 1u);

                    if (trainDataIndex < trainBufferSize) {
                        float roughness;
                        RGB diffuseReflectance, specularReflectance;
                        bsdf.getSurfaceParameters(
                            &diffuseReflectance, &specularReflectance, &roughness);

                        RadianceQuery radQuery;
                        createRadianceQuery(
                            positionInWorld, shadingFrame.normal, vOut,
                            roughness, diffuseReflectance, specularReflectance,
                            &radQuery);
                        plp.s->trainRadianceQueryBuffer[0][trainDataIndex] = radQuery;

                        TrainingVertexInfo vertInfo;
                        vertInfo.localThroughput = localThroughput;
                        vertInfo.prevVertexDataIndex = invalidVertexDataIndex;
                        vertInfo.pathLength = pathLength;
                        plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

                        // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
                        // EN: Initialize a target value by scattered radiance at the current vertex
                        //     by direct lighting (NEE).
                        plp.s->trainTargetBuffer[0][trainDataIndex] = directContNEE;
                        //if (!allFinite(directContNEE))
                        //    printf("NEE: (%g, %g, %g)\n",
                        //           directContNEE.x, directContNEE.y, directContNEE.z);
                    }
                    else {
                        trainDataIndex = invalidVertexDataIndex;
                    }
                }
            }
            else {
                (void)primaryPathSpread;
                (void)trainDataIndex;
            }
        }

        // Path extension loop
        PathTraceWriteOnlyPayload woPayload = {};
        PathTraceWriteOnlyPayload* woPayloadPtr = &woPayload;
        PathTraceReadWritePayload<useNRC> rwPayload = {};
        PathTraceReadWritePayload<useNRC>* rwPayloadPtr = &rwPayload;
        rwPayload.rng = rng;
        rwPayload.initImportance = initImportance;
        rwPayload.alpha = alpha;
        rwPayload.contribution = contribution;
        rwPayload.prevDirPDensity = dirPDensity;
        if constexpr (useNRC) {
            rwPayload.linearTileIndex = linearTileIndex;
            rwPayload.primaryPathSpread = primaryPathSpread;
            rwPayload.curSqrtPathSpread = 0.0f;
            rwPayload.prevLocalThroughput = localThroughput;
            rwPayload.prevTrainDataIndex = trainDataIndex;
            rwPayload.renderingPathEndsWithCache = false;
            rwPayload.isTrainingPath = isTrainingPath;
            rwPayload.isUnbiasedTrainingTile = isUnbiasedTrainingTile;
            rwPayload.trainingSuffixEndsWithCache = false;
        }
        rwPayload.pathLength = pathLength;
        Point3D rayOrg = positionInWorld;
        Vector3D rayDir = vIn;
        while (true) {
            const bool isValidSampling = rwPayload.prevDirPDensity > 0.0f && stc::isfinite(rwPayload.prevDirPDensity);
            if (!isValidSampling)
                break;

            ++rwPayload.pathLength;
            // JP: 通常のパストレーシングとNRCを正しく比較するには(特に通常のパストレーシングにおいて)
            //     反射回数制限を解除する必要がある。
            // EN: Disabling the limitation in the number of bounces (particularly for the base path tracing)
            //     is required to properly compare the base path tracing and NRC.
            if (rwPayload.pathLength >= plp.f->maxPathLength && plp.f->maxPathLength > 0)
                rwPayload.maxLengthTerminate = true;
            rwPayload.terminate = true;

            constexpr PathTracingRayType pathTraceRayType = useNRC ?
                PathTracingRayType::NRC : PathTracingRayType::Baseline;
            PathTraceRayPayloadSignature<useNRC>::trace(
                plp.f->travHandle, rayOrg.toNative(), rayDir.toNative(),
                0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                pathTraceRayType, maxNumRayTypes, pathTraceRayType,
                woPayloadPtr, rwPayloadPtr);
            if (rwPayload.terminate)
                break;
            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
        }
        contribution = rwPayload.contribution;

        plp.s->rngBuffer.write(launchIndex, rwPayload.rng);

        if constexpr (useNRC) {
            renderingPathEndsWithCache = rwPayload.renderingPathEndsWithCache;
            pathLength = rwPayload.pathLength;
            if (rwPayload.isTrainingPath && !rwPayload.trainingSuffixEndsWithCache) {
                TrainingSuffixTerminalInfo terminalInfo;
                terminalInfo.prevVertexDataIndex = rwPayload.prevTrainDataIndex;
                terminalInfo.hasQuery = false;
                terminalInfo.pathLength = rwPayload.pathLength;
                plp.s->trainSuffixTerminalInfoBuffer[rwPayload.linearTileIndex] = terminalInfo;
            }
        }
    }
    else {
        // JP: 環境光源を直接見ている場合の寄与を蓄積。
        // EN: Accumulate the contribution from the environmental light source directly seeing.
        if (useEnvLight) {
            const float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, bcB, bcC, 0.0f);
            const RGB luminance = plp.f->envLightPowerCoeff * RGB(getXYZ(texValue));
            contribution = luminance;
        }
    }

    if constexpr (useNRC) {
        const uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

        // JP: 無限遠にレイが飛んだか、ロシアンルーレットによってパストレースが完了したケース。
        // EN: When a ray goes infinity or the path ends with Russain roulette.
        if (!renderingPathEndsWithCache) {
            TerminalInfo terminalInfo;
            terminalInfo.alpha = RGB(0.0f, 0.0f, 0.0f);
            terminalInfo.pathLength = pathLength;
            terminalInfo.hasQuery = false;
            terminalInfo.isTrainingPixel = isTrainingPath;
            terminalInfo.isUnbiasedTile = isUnbiasedTrainingTile;
            plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;
        }

        plp.s->perFrameContributionBuffer[linearIndex] = contribution;
    }
    else {
        (void)renderingPathEndsWithCache;
        (void)pathLength;

        RGB prevColorResult = RGB(0.0f, 0.0f, 0.0f);
        if (plp.f->numAccumFrames > 0)
            prevColorResult = RGB(getXYZ(plp.s->beautyAccumBuffer.read(launchIndex)));
        const float curWeight = 1.0f / (1 + plp.f->numAccumFrames);
        const RGB colorResult = (1 - curWeight) * prevColorResult + curWeight * contribution;
        plp.s->beautyAccumBuffer.write(launchIndex, make_float4(colorResult.toNative(), 1.0f));
    }
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_closestHit_generic() {
    const uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    const uint32_t bufIdx = plp.f->bufferIndex;

    const auto sbtr = HitGroupSBTRecordData::get();
    const InstanceData &inst = plp.s->instanceDataBufferArray[bufIdx][optixGetInstanceId()];
    const GeometryInstanceData &geomInst = plp.s->geometryInstanceDataBuffer[sbtr.geomInstSlot];

    PathTraceWriteOnlyPayload* woPayload;
    PathTraceReadWritePayload<useNRC>* rwPayload;
    PathTraceRayPayloadSignature<useNRC>::get(&woPayload, &rwPayload);
    PCG32RNG &rng = rwPayload->rng;

    const Point3D rayOrigin(optixGetWorldRayOrigin());

    const auto hp = HitPointParameter::get();
    Point3D positionInWorld;
    Normal3D shadingNormalInWorld;
    Vector3D texCoord0DirInWorld;
    Normal3D geometricNormalInWorld;
    Point2D texCoord;
    float hypAreaPDensity;
    computeSurfacePoint<true, useSolidAngleSampling>(
        inst, geomInst, hp.primIndex, hp.bcB, hp.bcC,
        rayOrigin,
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);

    const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

    const Vector3D vOut = normalize(-Vector3D(optixGetWorldRayDirection()));
    const float frontHit = dot(vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
    if (plp.f->enableBumpMapping) {
        const Normal3D modLocalNormal = mat.readModifiedNormal(mat.normal, mat.normalDimInfo, texCoord, 0.0f);
        applyBumpMapping(modLocalNormal, &shadingFrame);
    }
    positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);
    const Vector3D vOutLocal = shadingFrame.toLocal(vOut);
    //if (!allFinite(vOutLocal)) {
    //    printf("(%g, %g, %g), (%g, %g, %g), (%g, %g, %g)\n",
    //           shadingFrame.tangent.x, shadingFrame.tangent.y, shadingFrame.tangent.z,
    //           shadingFrame.bitangent.x, shadingFrame.bitangent.y, shadingFrame.bitangent.z,
    //           shadingFrame.normal.x, shadingFrame.normal.y, shadingFrame.normal.z);
    //}

    const float dist2 = sqDistance(rayOrigin, positionInWorld);
    if constexpr (useNRC)
        rwPayload->curSqrtPathSpread += std::sqrt(dist2 / (rwPayload->prevDirPDensity * std::fabs(vOutLocal.z)));

    // Implicit Light Sampling
    if (vOutLocal.z > 0 && mat.emittance) {
        const float4 texValue = tex2DLod<float4>(mat.emittance, texCoord.x, texCoord.y, 0.0f);
        const RGB emittance(getXYZ(texValue));
        const float lightPDensity = hypAreaPDensity * dist2 / vOutLocal.z;
        const float bsdfPDensity = rwPayload->prevDirPDensity;
        const float misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
        const RGB directContImplicit = emittance * (misWeight / pi_v<float>);
        rwPayload->contribution += rwPayload->alpha * directContImplicit;

        if constexpr (useNRC) {
            // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
            // EN: Accumulate scattered radiance at the previous vertex by direct lighting (implicit)
            //     to the target value.
            if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex) {
                plp.s->trainTargetBuffer[0][rwPayload->prevTrainDataIndex] +=
                    rwPayload->prevLocalThroughput * directContImplicit;
                //if (!allFinite(rwPayload->prevLocalThroughput) ||
                //    !allFinite(directContImplicit))
                //    printf("Implicit: (%g, %g, %g), (%g, %g, %g)\n",
                //           rwPayload->prevLocalThroughput.x,
                //           rwPayload->prevLocalThroughput.y,
                //           rwPayload->prevLocalThroughput.z,
                //           directContImplicit.x,
                //           directContImplicit.y,
                //           directContImplicit.z);
            }
        }
    }

    // Russian roulette
    bool performRR = true;
    bool terminatedByRR = false;
    float recContinueProb = 1.0f;
    if constexpr (useNRC) {
        if (rwPayload->isTrainingPath)
            performRR = rwPayload->pathLength > 2;
    }
    if (performRR) {
        float continueProb = std::fmin(sRGB_calcLuminance(rwPayload->alpha) / rwPayload->initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb || rwPayload->maxLengthTerminate) {
            if constexpr (useNRC) {
                if (rwPayload->renderingPathEndsWithCache &&
                    rwPayload->isTrainingPath && rwPayload->isUnbiasedTrainingTile)
                    return;
                terminatedByRR = true;
            }
            else {
                return;
            }
        }
        recContinueProb = 1.0f / continueProb;
    }

    BSDF bsdf;
    bsdf.setup(mat, texCoord, 0.0f);

    if constexpr (useNRC) {
        bool endsWithCache = false;
        const bool pathIsSpreadEnough =
            pow2(rwPayload->curSqrtPathSpread) > pathTerminationFactor * rwPayload->primaryPathSpread;
        endsWithCache |= pathIsSpreadEnough;
        if (rwPayload->renderingPathEndsWithCache &&
            rwPayload->isTrainingPath && rwPayload->isUnbiasedTrainingTile)
            endsWithCache = false;

        if (endsWithCache) {
            const uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;

            float roughness;
            RGB diffuseReflectance, specularReflectance;
            bsdf.getSurfaceParameters(
                &diffuseReflectance, &specularReflectance, &roughness);

            // JP: Radianceクエリーのための情報を記録する。
            // EN: Store information for radiance query.
            RadianceQuery radQuery;
            createRadianceQuery(
                positionInWorld, shadingFrame.normal, vOut,
                roughness, diffuseReflectance, specularReflectance,
                &radQuery);

            if (!rwPayload->renderingPathEndsWithCache) {
                plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;

                TerminalInfo terminalInfo;
                terminalInfo.alpha = rwPayload->alpha;
                terminalInfo.pathLength = rwPayload->pathLength;
                terminalInfo.hasQuery = true;
                terminalInfo.isTrainingPixel = rwPayload->isTrainingPath;
                terminalInfo.isUnbiasedTile = rwPayload->isUnbiasedTrainingTile;
                plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;

                rwPayload->renderingPathEndsWithCache = true;
                if (rwPayload->isTrainingPath)
                    rwPayload->curSqrtPathSpread = 0;
                else
                    return;
            }
            else {
                // JP: 訓練データバッファーがフルの場合は既にTraining Suffixは終了したことになっている。
                // EN: The training suffix should have been ended if the training data buffer is full.
                if (!rwPayload->trainingSuffixEndsWithCache) {
                    uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
                    plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

                    // JP: 直前のTraining VertexへのリンクとともにTraining Suffixを終了させる。
                    // EN: Finish the training suffix with the link to the previous training vertex.
                    TrainingSuffixTerminalInfo terminalInfo;
                    terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                    terminalInfo.hasQuery = true;
                    terminalInfo.pathLength = rwPayload->pathLength;
                    plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

                    rwPayload->trainingSuffixEndsWithCache = true;
                }
                return;
            }
        }
    }

    if constexpr (useNRC) {
        if (terminatedByRR)
            return;
    }
    rwPayload->alpha *= recContinueProb;
    if constexpr (useNRC) {
        if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex)
            plp.s->trainVertexInfoBuffer[rwPayload->prevTrainDataIndex].localThroughput *= recContinueProb;
    }

    // Next Event Estimation (Explicit Light Sampling)
    const RGB directContNEE = performNextEventEstimation(
        positionInWorld, vOutLocal, shadingFrame, bsdf, rng);
    rwPayload->contribution += rwPayload->alpha * directContNEE;

    // generate a next ray.
    Vector3D vInLocal;
    float dirPDensity;
    const RGB localThroughput = bsdf.sampleThroughput(
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    rwPayload->alpha *= localThroughput;
    const Vector3D vIn = shadingFrame.fromLocal(vInLocal);

    woPayload->nextOrigin = positionInWorld;
    woPayload->nextDirection = vIn;
    rwPayload->prevDirPDensity = dirPDensity;
    if constexpr (useNRC)
        rwPayload->prevLocalThroughput = localThroughput;
    rwPayload->terminate = false;

    if constexpr (useNRC) {
        // JP: 訓練データエントリーの確保。
        // EN: Allocate a training data entry.
        if (rwPayload->isTrainingPath && !rwPayload->trainingSuffixEndsWithCache) {
            const uint32_t trainDataIndex = atomicAdd(plp.s->numTrainingData[bufIdx], 1u);

            // TODO?: 訓練データ数の正確な推定のためにtrainingSuffixEndsWithCacheのチェックをここに持ってくる？

            float roughness;
            RGB diffuseReflectance, specularReflectance;
            bsdf.getSurfaceParameters(
                &diffuseReflectance, &specularReflectance, &roughness);

            RadianceQuery radQuery;
            createRadianceQuery(
                positionInWorld, shadingFrame.normal, vOut,
                roughness, diffuseReflectance, specularReflectance,
                &radQuery);

            if (trainDataIndex < trainBufferSize) {
                plp.s->trainRadianceQueryBuffer[0][trainDataIndex] = radQuery;

                // JP: ローカルスループットと前のTraining Vertexへのリンクを記録。
                // EN: Record the local throughput and the link to the previous training vertex.
                TrainingVertexInfo vertInfo;
                vertInfo.localThroughput = localThroughput;
                vertInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                vertInfo.pathLength = rwPayload->pathLength;
                plp.s->trainVertexInfoBuffer[trainDataIndex] = vertInfo;

                // JP: 現在の頂点に対する直接照明(NEE)によるScattered Radianceでターゲット値を初期化。
                // EN: Initialize a target value by scattered radiance at the current vertex by
                //     direct lighting (NEE).
                plp.s->trainTargetBuffer[0][trainDataIndex] = directContNEE;
                //if (!allFinite(directContNEE))
                //    printf("NEE: (%g, %g, %g)\n",
                //           directContNEE.x, directContNEE.y, directContNEE.z);

                rwPayload->prevTrainDataIndex = trainDataIndex;
            }
            // JP: 訓練データがバッファーを溢れた場合は強制的にTraining Suffixを終了させる。
            // EN: Forcefully end the training suffix if the training data buffer become full.
            else {
                const uint32_t offset = plp.s->imageSize.x * plp.s->imageSize.y;
                plp.s->inferenceRadianceQueryBuffer[offset + rwPayload->linearTileIndex] = radQuery;

                TrainingSuffixTerminalInfo terminalInfo;
                terminalInfo.prevVertexDataIndex = rwPayload->prevTrainDataIndex;
                terminalInfo.hasQuery = true;
                terminalInfo.pathLength = rwPayload->pathLength;
                plp.s->trainSuffixTerminalInfoBuffer[rwPayload->linearTileIndex] = terminalInfo;

                rwPayload->trainingSuffixEndsWithCache = true;
            }
        }
    }
}

template <bool useNRC>
CUDA_DEVICE_FUNCTION CUDA_INLINE void pathTrace_miss_generic() {
    if (!plp.s->envLightTexture || !plp.f->enableEnvLight)
        return;

    PathTraceReadWritePayload<useNRC>* rwPayload;
    PathTraceRayPayloadSignature<useNRC>::get(nullptr, &rwPayload);

    const Vector3D rayDir = normalize(Vector3D(optixGetWorldRayDirection()));
    float posPhi, theta;
    toPolarYUp(rayDir, &posPhi, &theta);

    float phi = posPhi + plp.f->envLightRotation;
    phi = phi - floorf(phi / (2 * pi_v<float>)) * 2 * pi_v<float>;
    const Point2D texCoord(phi / (2 * pi_v<float>), theta / pi_v<float>);

    // Implicit Light Sampling
    const float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, texCoord.x, texCoord.y, 0.0f);
    const RGB luminance = plp.f->envLightPowerCoeff * RGB(getXYZ(texValue));
    const float uvPDF = plp.s->envLightImportanceMap.evaluatePDF(texCoord.x, texCoord.y);
    const float hypAreaPDensity = uvPDF / (2 * pi_v<float> * pi_v<float> * std::sin(theta));
    const float lightPDensity = probToSampleEnvLight * hypAreaPDensity;
    const float bsdfPDensity = rwPayload->prevDirPDensity;
    const float misWeight = pow2(bsdfPDensity) / (pow2(bsdfPDensity) + pow2(lightPDensity));
    const RGB directContImplicit = misWeight * luminance;
    rwPayload->contribution += rwPayload->alpha * directContImplicit;

    if constexpr (useNRC) {
        // JP: 1つ前の頂点に対する直接照明(Implicit)によるScattered Radianceをターゲット値に加算。
        // EN: Accumulate scattered radiance at the previous vertex by direct lighting (implicit)
        //     to the target value.
        if (rwPayload->isTrainingPath && rwPayload->prevTrainDataIndex != invalidVertexDataIndex) {
            plp.s->trainTargetBuffer[0][rwPayload->prevTrainDataIndex] +=
                rwPayload->prevLocalThroughput * directContImplicit;
            //if (!allFinite(rwPayload->prevLocalThroughput) ||
            //    !allFinite(directContImplicit))
            //    printf("Implicit: (%g, %g, %g), (%g, %g, %g)\n",
            //           rwPayload->prevLocalThroughput.x,
            //           rwPayload->prevLocalThroughput.y,
            //           rwPayload->prevLocalThroughput.z,
            //           directContImplicit.x,
            //           directContImplicit.y,
            //           directContImplicit.z);
        }
    }
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceBaseline)() {
    pathTrace_raygen_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceBaseline)() {
    pathTrace_closestHit_generic<false>();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTraceBaseline)() {
    pathTrace_miss_generic<false>();
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(pathTraceNRC)() {
    pathTrace_raygen_generic<true>();
}

CUDA_DEVICE_KERNEL void RT_CH_NAME(pathTraceNRC)() {
    pathTrace_closestHit_generic<true>();
}

CUDA_DEVICE_KERNEL void RT_MS_NAME(pathTraceNRC)() {
    pathTrace_miss_generic<true>();
}



CUDA_DEVICE_KERNEL void RT_RG_NAME(visualizePrediction)() {
    const uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    const uint32_t linearIndex = launchIndex.y * plp.s->imageSize.x + launchIndex.x;
    const uint32_t bufIdx = plp.f->bufferIndex;

    const GBuffer0Elements gb0Elems = plp.s->GBuffer0[bufIdx].read(launchIndex);
    const uint32_t instSlot = gb0Elems.instSlot;
    const float bcB = decodeBarycentric(gb0Elems.qbcB);
    const float bcC = decodeBarycentric(gb0Elems.qbcC);

    const PerspectiveCamera &camera = plp.f->camera;

    if (instSlot != 0xFFFFFFFF) {
        const uint32_t geomInstSlot = gb0Elems.geomInstSlot;
        const InstanceData &inst = plp.s->instanceDataBufferArray[bufIdx][instSlot];
        const GeometryInstanceData &geomInst = plp.s->geometryInstanceDataBuffer[geomInstSlot];
        Point3D positionInWorld;
        Normal3D geometricNormalInWorld;
        Normal3D shadingNormalInWorld;
        Vector3D texCoord0DirInWorld;
        Point2D texCoord;
        computeSurfacePoint(
            inst, geomInst,
            gb0Elems.primIndex, bcB, bcC,
            &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
            &geometricNormalInWorld, &texCoord);

        const MaterialData &mat = plp.s->materialDataBuffer[geomInst.materialSlot];

        Vector3D vOut = camera.position - positionInWorld;
        float primaryDist2 = vOut.sqLength();
        vOut /= std::sqrt(primaryDist2);
        const float primaryDotVN = dot(vOut, geometricNormalInWorld);
        const float frontHit = primaryDotVN >= 0.0f ? 1.0f : -1.0f;
        // Offsetting assumes BRDF.
        positionInWorld = offsetRayOrigin(positionInWorld, frontHit * geometricNormalInWorld);

        ReferenceFrame shadingFrame(shadingNormalInWorld, texCoord0DirInWorld);
        if (plp.f->enableBumpMapping) {
            const Normal3D modLocalNormal = mat.readModifiedNormal(mat.normal, mat.normalDimInfo, texCoord, 0.0f);
            applyBumpMapping(modLocalNormal, &shadingFrame);
        }

        BSDF bsdf;
        bsdf.setup(mat, texCoord, 0.0f);

        float roughness;
        RGB diffuseReflectance, specularReflectance;
        bsdf.getSurfaceParameters(
            &diffuseReflectance, &specularReflectance, &roughness);

        RadianceQuery radQuery;
        createRadianceQuery(
            positionInWorld, shadingFrame.normal, vOut,
            roughness, diffuseReflectance, specularReflectance,
            &radQuery);

        plp.s->inferenceRadianceQueryBuffer[linearIndex] = radQuery;
    }
    else {
        //// JP: 環境光源を直接見ている場合の寄与を蓄積。
        //// EN: Accumulate the contribution from the environmental light source directly seeing.
        //if (useEnvLight) {
        //    const float4 texValue = tex2DLod<float4>(plp.s->envLightTexture, bcB, bcC, 0.0f);
        //    const RGB luminance = plp.f->envLightPowerCoeff * RGB(getXYZ(texValue));
        //    contribution = luminance;
        //}
    }

    TerminalInfo terminalInfo;
    terminalInfo.alpha = RGB(1.0f);
    terminalInfo.pathLength = 1;
    terminalInfo.hasQuery = instSlot != 0xFFFFFFFF;
    terminalInfo.isTrainingPixel = false;
    terminalInfo.isUnbiasedTile = false;
    plp.s->inferenceTerminalInfoBuffer[linearIndex] = terminalInfo;
}
